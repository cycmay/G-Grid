//
// Created by bicycle on 2020/11/27.
//

#include "Index.h"
#include <hip/hip_runtime.h>
#include <chrono>

void Index::updateMessage(const Message& m, GraphGrid& G) {
// input : a message m and the G-Grid
// output: updated G-Grid
    int c = G.getCell(m.edge);
    G.cells[c].messageList.append(m);
    if(c != getCellFromOT(m.object)){
        int c_ = getCellFromOT(m.object);
        G.cells[c_].messageList.append(Message(m.object, Edge(), 0.0, m.t));
    }
    setOT(m.object, LatestLocation(c, m.edge, m.dist));

}

int Index::getCellFromOT(int o) {
    return objectTable.table[o].cid;
}

void Index::setOT(int o, LatestLocation l) {
    objectTable.table[o] = l;
}

void Index::messageClean(std::vector<MessageList> ML) {
// input : a set of message lists L
// output: up-to-date object locations R
    for(auto & itr : ML){
        // append a new bucket ζ to the tail of L; point p l to ζ
        itr.pl = itr.addBucket();
        // clean out-time bucket
        for(auto bkt=itr.ph->next, bktPrior=itr.ph;;){
            if(bkt!= nullptr){
                if((std::chrono::high_resolution_clock::now()-bkt->t).count()>timeout){
                    bktPrior->next=bkt->next;
                    itr.deleteBkt(bkt);
                    bkt=bktPrior->next;
                }
                else {
                    bktPrior = bkt;
                    bkt = bkt->next;
                }
            }
            else{
                break;
            }
        }
        // create a size L.n array L.A on the GPU
        
        hipMalloc((void))
        // copy buckets before p l for all L ∈ L into L.A
    }
}

void Index::setTimeout(int t) {
    timeout=t;
}

int Index::getTimeout(void) {
    return timeout;
}
